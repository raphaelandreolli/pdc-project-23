/*
 * Simplified simulation of high-energy particle storms
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2017/2018
 *
 * Version: 2.0
 *
 * Code prepared to be used with the Tablon on-line judge.
 * The current Parallel Computing course includes contests using:
 * OpenMP, MPI, and CUDA.
 *
 * (c) 2018 Arturo Gonzalez-Escribano, Eduardo Rodriguez-Gutiez
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


/* Use fopen function in local tests. The Tablon online judge software
   substitutes it by a different function to run in its sandbox */
#ifdef CP_TABLON
#include "cputilstablon.h"
#else
#define    cp_open_file(name) fopen(name,"r")
#endif

/* Function to get wall time */
double cp_Wtime(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}


#define THRESHOLD    0.001f
#define RAD 1
#define TPB 256
#define RAND_MIN -1.0

/* Structure used to store data for one storm of particles */
typedef struct {
    int size;    // Number of particles
    int *posval; // Positions and values
} Storm;



/* Kernel Function to update energy due to particle bombardment on cells */
__global__ void kernelupdate( float *d_layer,float *h_layer, int layer_size, Storm storms) {

//global thread index
    int k = blockIdx.x * blockDim.x + threadIdx.x;

//local thread index for local shared array
	const int s_idx = threadIdx.x + RAD;
  
//dynamic shared memory
	extern __shared__ float s_in[];

    if (k >= layer_size) return;



	float energy_value = 0.0f ;
	for (int i=0; i<storms.size; i++)
	{
 /* 1. Compute the absolute value of the distance between the impact position and the k-th position of the layer */
		float energy = storms.posval[i*2+1] * 1000;
		int distance = abs(storms.posval[i*2] - k) + 1;
		float attenuation = sqrtf(static_cast<float>(distance));
     /*  Compute attenuated energy */
		float energy_k = energy / layer_size / attenuation;


		if (energy_k >= THRESHOLD / layer_size || energy_k <= -THRESHOLD / layer_size)
		{
			energy_value +=energy_k;
	    }
	}

  /*  Add intermediate impacts energies before relaxation to  cells */
	h_layer[k]=h_layer[k]+energy_value;
	__syncthreads();

//copying data from global to local shared array
	s_in[s_idx] = h_layer[k];
	if (threadIdx.x < RAD) {
		if (k==0){
			s_in[s_idx - RAD] = h_layer[k];
			}
		else {
				s_in[s_idx - RAD] = h_layer[k - RAD];
				}
		s_in[s_idx + blockDim.x] = h_layer[k + blockDim.x];
	}
	__syncthreads();

   /*  Energy relaxation between storms, preserving first and last element */
    if ( k == 0 || k == layer_size-1 ) {
		d_layer[k]=h_layer[k];
    }
	else {
		d_layer[k] = (s_in[s_idx-1]+ s_in[s_idx] + s_in[s_idx+1])/3;
	}
}


/* ANCILLARY FUNCTIONS: These are not called from the code section which is measured, leave untouched */
/* DEBUG function: Prints the layer status */
void debug_print(int layer_size, float *layer, int *positions, float *maximum, int num_storms ) {
    int i,k;
    /* Only print for array size up to 35 (change it for bigger sizes if needed) */
    if ( layer_size <= 35 ) {
        /* Traverse layer */
        for( k=0; k<layer_size; k++ ) {
            /* Print the energy value of the current cell */
            printf("%10.4f |", layer[k] );

            /* Compute the number of characters.
               This number is normalized, the maximum level is depicted with 60 characters */
            int ticks = (int)( 60 * layer[k] / maximum[num_storms-1] );

            /* Print all characters except the last one */
            for (i=0; i<ticks-1; i++ ) printf("o");

            /* If the cell is a local maximum print a special trailing character */
            if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                printf("x");
            else
                printf("o");

            /* If the cell is the maximum of any storm, print the storm mark */
            for (i=0; i<num_storms; i++)
                if ( positions[i] == k ) printf(" M%d", i );

            /* Line feed */
            printf("\n");
        }
    }
}

/*
 * Function: Read data of particle storms from a file
 */
Storm read_storm_file( char *fname ) {
    FILE *fstorm = cp_open_file( fname );
    if ( fstorm == NULL ) {
        fprintf(stderr,"Error: Opening storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    Storm storm;
    int ok = fscanf(fstorm, "%d", &(storm.size) );
    if ( ok != 1 ) {
        fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
    if ( storm.posval == NULL ) {
        fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
        exit( EXIT_FAILURE );
    }

    int elem;
    for ( elem=0; elem<storm.size; elem++ ) {
        ok = fscanf(fstorm, "%d %d\n",
                    &(storm.posval[elem*2]),
                    &(storm.posval[elem*2+1]) );
        if ( ok != 2 ) {
            fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
            exit( EXIT_FAILURE );
        }
    }
    fclose( fstorm );

    return storm;
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
    int i,k;

    /* 1.1. Read arguments */
    if (argc<3) {
        fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
        exit( EXIT_FAILURE );
    }

    int layer_size = atoi( argv[1] );
    int num_storms = argc-2;
    int numBlocks = (atoi( argv[1] ) + TPB - 1) / TPB; // Number of blocks needed

    Storm storms[ num_storms ];

    /* 1.2. Read storms information */
    for( i=2; i<argc; i++ )
        storms[i-2] = read_storm_file( argv[i] );

    /* 1.3. Intialize maximum levels to zero */
    float maximum[ num_storms ];
    int positions[ num_storms ];
    for (i=0; i<num_storms; i++) {
        maximum[i] = 0.0f;
        positions[i] = 0;
    }

    /* 2. Begin time measurement */
	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    double ttotal = cp_Wtime();

    /* START: Do NOT optimize/parallelize the code of the main program above this point */

    /* 3. Allocate memory for the layer and initialize to zero */
    float *layer = (float *)malloc( sizeof(float) * layer_size );

    if ( layer == NULL  ) {
        fprintf(stderr,"Error: Allocating the layer memory\n");
        exit( EXIT_FAILURE );
    }
    for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;


    /* 4. Storms simulation */
    for( i=0; i<num_storms; i++) {

 // Load storm to device memory
	    Storm storm;
	    storm.size= storms[i].size;
	    size_t size1=2*storms[i].size* sizeof(int);
	    hipMalloc(&storm.posval, size1);
        hipMemcpy(storm.posval, storms[i].posval, size1,
               hipMemcpyHostToDevice);

// Allocate layer and intermediate layer (h_layer) before relaxation in device memory
	    float *d_layer;
		hipMalloc( &d_layer,sizeof(float) * layer_size );

	    float *h_layer;
		hipMalloc( &h_layer,sizeof(float) * layer_size );

// Copy layer from host CPU to device GPU
		hipMemcpy(h_layer, layer, layer_size * sizeof(float), hipMemcpyHostToDevice);

 
        dim3 blockDim(TPB);
        dim3 gridDim(numBlocks);

//Size for shared memory array        
        const size_t smemSize = (blockDim.x + 2 * RAD) * sizeof(float);

// Invoke kernel
		kernelupdate<<<gridDim, blockDim,smemSize>>>(d_layer, h_layer, layer_size, storm);
        hipDeviceSynchronize() ;

// Read layer from device memory
		size_t size=sizeof(float) * layer_size;
	    hipMemcpy(layer, d_layer, size,
               hipMemcpyDeviceToHost);

// Free device memory
    hipFree(h_layer);
		hipFree(d_layer);

        /* 4.3. Locate the maximum value in the layer, and its position */
        for( k=1; k<layer_size-1; k++ ) {
            /* Check it only if it is a local maximum */
            if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
                if ( layer[k] > maximum[i] ) {
                    maximum[i] = layer[k];
                    positions[i] = k;
                }
            }
        }
    }

    /* END: Do NOT optimize/parallelize the code below this point */

    /* 5. End time measurement */
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    ttotal = cp_Wtime() - ttotal;

    /* 6. DEBUG: Plot the result (only for layers up to 35 points) */
    #ifdef DEBUG
    debug_print( layer_size, layer, positions, maximum, num_storms );
    #endif

    /* 7. Results output, used by the Tablon online judge software */
    printf("\n");
    /* 7.1. Total computation time */
    printf("Time: %lf\n", ttotal );
    /* 7.2. Print the maximum levels */
    printf("Result:");
    for (i=0; i<num_storms; i++)
        printf(" %d %f", positions[i], maximum[i] );
    printf("\n");

    /* 8. Free resources */
    for( i=0; i<argc-2; i++ )
        free( storms[i].posval );

    /* 9. Program ended successfully */
    return 0;
}

/*
 * Simplified simulation of high-energy particle storms
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2017/2018
 *
 * Version: 2.0
 *
 * Code prepared to be used with the Tablon on-line judge.
 * The current Parallel Computing course includes contests using:
 * OpenMP, MPI, and CUDA.
 *
 * (c) 2018 Arturo Gonzalez-Escribano, Eduardo Rodriguez-Gutiez
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


/* Use fopen function in local tests. The Tablon online judge software 
   substitutes it by a different function to run in its sandbox */
#ifdef CP_TABLON
#include "cputilstablon.h"
#else
#define    cp_open_file(name) fopen(name,"r")
#endif

/* Function to get wall time */
double cp_Wtime(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}


#define THRESHOLD    0.001f
#define RAD 1
#define TPB 256
#define RAND_MIN -1.0

/* Structure used to store data for one storm of particles */
typedef struct {
    int size;    // Number of particles
    int *posval; // Positions and values
} Storm;



/* THIS FUNCTION CAN BE MODIFIED */
/* Function to update a single position of the layer */
__global__ void kernelupdate( float *d_layer,float *h_layer, int layer_size, Storm storms,float* maxresult_block, int* maxindex_block) {
    /* 1. Compute the absolute value of the distance between the
        impact position and the k-th position of the layer */
		
    int k = blockIdx.x * blockDim.x + threadIdx.x;
		
	
	extern __shared__ float s_in[];

    __shared__ float maxVals[TPB];
    __shared__ int maxIndices[TPB];
	
	const int s_idx = threadIdx.x + RAD;
	int tid = threadIdx.x;

    if (k >= layer_size) return;
	

	
	float energy_value = 0.0f ;
	for (int i=0; i<storms.size; i++)
	{
		float energy = storms.posval[i*2+1] * 1000;
		int distance = abs(storms.posval[i*2] - k) + 1;
		float attenuation = sqrtf(static_cast<float>(distance));
		float energy_k = energy / layer_size / attenuation;


		if (energy_k >= THRESHOLD / layer_size || energy_k <= -THRESHOLD / layer_size) 
		{
			energy_value +=energy_k; 
	    }
	}
	h_layer[k]=h_layer[k]+energy_value;
	__syncthreads();

	
	s_in[s_idx] = h_layer[k];		
	if (threadIdx.x < RAD) {
		if (k==0){
			s_in[s_idx - RAD] = h_layer[k];
			}
		else {
				s_in[s_idx - RAD] = h_layer[k - RAD];
				}
		s_in[s_idx + blockDim.x] = h_layer[k + blockDim.x];
	}
	__syncthreads();
    if ( k == 0 || k == layer_size-1 ) {
		d_layer[k]=h_layer[k];	
    }
	else {
		d_layer[k] = (s_in[s_idx-1]+ s_in[s_idx] + s_in[s_idx+1])/3;
	}
	__syncthreads();
	
    float maxValue = RAND_MIN;
    int maxIdx = -1;
	
    while (k < layer_size) {
        float val = d_layer[k];
        if (val > maxValue) {
            maxValue = val;
            maxIdx = k;
        }
        k += blockDim.x * gridDim.x;
    }

    maxVals[tid] = maxValue;
    maxIndices[tid] = maxIdx;

    __syncthreads();
	
    // Perform parallel reduction to find the maximum value and its index
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            if (maxVals[tid] < maxVals[tid + stride]) {
                maxVals[tid] = maxVals[tid + stride];
                maxIndices[tid] = maxIndices[tid + stride];
            }
        }
        __syncthreads();
    }
    
    // Store the final result in global memory
    if (tid == 0) {
        maxresult_block[blockIdx.x] = maxVals[0];
        maxindex_block[blockIdx.x] = maxIndices[0];
    }
}


/* ANCILLARY FUNCTIONS: These are not called from the code section which is measured, leave untouched */
/* DEBUG function: Prints the layer status */
void debug_print(int layer_size, float *layer, int *positions, float *maximum, int num_storms ) {
    int i,k;
    /* Only print for array size up to 35 (change it for bigger sizes if needed) */
    if ( layer_size <= 35 ) {
        /* Traverse layer */
        for( k=0; k<layer_size; k++ ) {
            /* Print the energy value of the current cell */
            printf("%10.4f |", layer[k] );

            /* Compute the number of characters. 
               This number is normalized, the maximum level is depicted with 60 characters */
            int ticks = (int)( 60 * layer[k] / maximum[num_storms-1] );

            /* Print all characters except the last one */
            for (i=0; i<ticks-1; i++ ) printf("o");

            /* If the cell is a local maximum print a special trailing character */
            if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                printf("x");
            else
                printf("o");

            /* If the cell is the maximum of any storm, print the storm mark */
            for (i=0; i<num_storms; i++) 
                if ( positions[i] == k ) printf(" M%d", i );

            /* Line feed */
            printf("\n");
        }
    }
}

/*
 * Function: Read data of particle storms from a file
 */
Storm read_storm_file( char *fname ) {
    FILE *fstorm = cp_open_file( fname );
    if ( fstorm == NULL ) {
        fprintf(stderr,"Error: Opening storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    Storm storm;    
    int ok = fscanf(fstorm, "%d", &(storm.size) );
    if ( ok != 1 ) {
        fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
    if ( storm.posval == NULL ) {
        fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
        exit( EXIT_FAILURE );
    }
    
    int elem;
    for ( elem=0; elem<storm.size; elem++ ) {
        ok = fscanf(fstorm, "%d %d\n", 
                    &(storm.posval[elem*2]),
                    &(storm.posval[elem*2+1]) );
        if ( ok != 2 ) {
            fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
            exit( EXIT_FAILURE );
        }
    }
    fclose( fstorm );

    return storm;
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
    int i,k;



    /* 1.1. Read arguments */
    if (argc<3) {
        fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
        exit( EXIT_FAILURE );
    }

    int layer_size = atoi( argv[1] );
    int num_storms = argc-2;
	
	int numBlocks = (atoi( argv[1] ) + TPB - 1) / TPB; // Number of blocks needed
    Storm storms[ num_storms ];

    /* 1.2. Read storms information */
    for( i=2; i<argc; i++ ) 
        storms[i-2] = read_storm_file( argv[i] );

    /* 1.3. Intialize maximum levels to zero */
    float maximum[ num_storms ];
    int positions[ num_storms ];
    for (i=0; i<num_storms; i++) {
        maximum[i] = 0.0f;
        positions[i] = 0;
    }

    /* 2. Begin time measurement */
	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    double ttotal = cp_Wtime();

    /* START: Do NOT optimize/parallelize the code of the main program above this point */

    /* 3. Allocate memory for the layer and initialize to zero */
    float *layer = (float *)malloc( sizeof(float) * layer_size );

    if ( layer == NULL  ) {
        fprintf(stderr,"Error: Allocating the layer memory\n");
        exit( EXIT_FAILURE );
    }
    for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;

    
    /* 4. Storms simulation */
    for( i=0; i<num_storms; i++) {

	    Storm storm;
	    storm.size= storms[i].size;
	    size_t size1=2*storms[i].size* sizeof(int);
	    hipMalloc(&storm.posval, size1);
        hipMemcpy(storm.posval, storms[i].posval, size1,
               hipMemcpyHostToDevice);        

	    float *d_layer;
		hipMalloc( &d_layer,sizeof(float) * layer_size );
			
	    float *h_layer;
		hipMalloc( &h_layer,sizeof(float) * layer_size );	

		hipMemcpy(h_layer, layer, layer_size * sizeof(float), hipMemcpyHostToDevice);	
		
    // Allocate device memory for the result and index arrays
		float* d_maxresult_block;
		int* d_maxindex_block;
		hipMalloc((void**)&d_maxresult_block, numBlocks * sizeof(float));
		hipMalloc((void**)&d_maxindex_block, numBlocks * sizeof(int));
		
         // Invoke kernel
        dim3 blockDim(TPB);
        dim3 gridDim(numBlocks);
        const size_t smemSize = (blockDim.x + 2 * RAD) * sizeof(float);
		kernelupdate<<<gridDim, blockDim,smemSize>>>(d_layer, h_layer, layer_size, storm, d_maxresult_block,d_maxindex_block);
        hipDeviceSynchronize() ;

		size_t size=sizeof(float) * layer_size;
	    hipMemcpy(layer, d_layer, size,
               hipMemcpyDeviceToHost);
        

		
    // Copy the results from device to host
		float* h_result = (float*)malloc(numBlocks * sizeof(float));
		int* h_index = (int*)malloc(numBlocks * sizeof(int));
		hipMemcpy(h_result, d_maxresult_block, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_index, d_maxindex_block, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
	
    // Find the maximum value and its index among the block-wise results

		for (int j = 0; j < numBlocks; j++) {
			if (h_result[j] > maximum[i]) {
				maximum[i] = h_result[j];
				positions[i] = h_index[j];
			}
		}
		
	// Free allocated memory
		free(h_result);
		free(h_index);
		hipFree(h_layer);
		hipFree(d_layer);
		hipFree(d_maxresult_block);
		hipFree(d_maxindex_block);
    }

    /* END: Do NOT optimize/parallelize the code below this point */

    /* 5. End time measurement */
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    ttotal = cp_Wtime() - ttotal;

    /* 6. DEBUG: Plot the result (only for layers up to 35 points) */
    #ifdef DEBUG
    debug_print( layer_size, layer, positions, maximum, num_storms );
    #endif

    /* 7. Results output, used by the Tablon online judge software */
    printf("\n");
    /* 7.1. Total computation time */
    printf("Time: %lf\n", ttotal );
    /* 7.2. Print the maximum levels */
    printf("Result:");
    for (i=0; i<num_storms; i++)
        printf(" %d %f", positions[i], maximum[i] );
    printf("\n");

    /* 8. Free resources */    
    for( i=0; i<argc-2; i++ )
        free( storms[i].posval );

    /* 9. Program ended successfully */
    return 0;
}
